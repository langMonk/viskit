#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <ivhd/IGraph.h>
#include <ivhd/IParticleSystem.h>
#include "constants.h"
#include "caster/caster_cuda.cuh"
#include <math.h>

// initialize pos in Samples
// initialize num_components
__global__ void initializeSamples(int n, Sample* samples, float2* positions, short* sampleFreq)
{
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < n) 
	{
		Sample sample;
		sample.pos = positions[i];
		sample.v = { 0, 0 };
		sample.num_components = sampleFreq[i];
		// FIXME - malloc can return NULL
		sample.components =
			static_cast<float2*>(malloc(sample.num_components * sizeof(float2)));
		samples[i] = sample;
	}
}

__global__ void initializeDistances(int nDst, DistElem* distances, short2* dstIndexes, Sample* samples)
{
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < nDst) 
	{
		DistElem dst = distances[i];
		dst.comp1 = &samples[dst.i].components[dstIndexes[i].x];
		dst.comp2 = &samples[dst.j].components[dstIndexes[i].y];
		distances[i] = dst;
	}
}

void CasterCuda::initializeHelperVectors()
{
	/*
	 * calculate number of distances for each sample and index of each distance
	 * for a given sample
	 */
	short* sampleFreq = new short[positions.size()];
	for (unsigned i = 0; i < positions.size(); i++) 
	{
		sampleFreq[i] = 0;
	}

	short2* dstIndexes = new short2[distances.size()];

	for (unsigned i = 0; i < distances.size(); i++) 
	{
		dstIndexes[i] = { sampleFreq[distances[i].i]++,
		  sampleFreq[distances[i].j]++ };
	}

	// initialize samples
	short* d_sample_freq;
	cuCall(hipMalloc(&d_sample_freq, positions.size() * sizeof(short)));
	cuCall(hipMemcpy(d_sample_freq, sampleFreq, sizeof(short) * positions.size(),
		hipMemcpyHostToDevice));

	delete sampleFreq;

	initializeSamples<<<positions.size() / 256 + 1, 256>>> (
		positions.size(), d_samples, d_positions, d_sample_freq);
	cuCall(hipFree(d_sample_freq));

	// initialize comps in Distances in device memory
	short2* d_dst_indexes;
	cuCall(hipMalloc(&d_dst_indexes, distances.size() * sizeof(short2)));
	cuCall(hipMemcpy(d_dst_indexes, dstIndexes,
		sizeof(short2) * distances.size(), hipMemcpyHostToDevice));

	initializeDistances <<<distances.size() / 256 + 1, 256 >>> (
		distances.size(), d_distances, d_dst_indexes, d_samples);
	cuCall(hipFree(d_dst_indexes));
	delete dstIndexes;
}

bool CasterCuda::allocateInitializeDeviceMemory()
{
	cuCall(hipMalloc(&d_positions, positions.size() * sizeof(float2)));
	cuCall(hipMalloc(&d_samples, positions.size() * sizeof(Sample)));
	cuCall(hipMalloc(&d_distances, distances.size() * sizeof(DistElem)));
	cuCall(hipMalloc(&d_errors, distances.size() * sizeof(float)));

	cuCall(hipMemcpy(d_positions, &positions[0],
		sizeof(float2) * positions.size(), hipMemcpyHostToDevice));
	cuCall(hipMemset(d_samples, 0, positions.size() * sizeof(Sample)));
	cuCall(hipMemset(d_errors, 0, distances.size() * sizeof(float)));
	cuCall(hipMemcpy(d_distances, &distances[0],
		sizeof(DistElem) * distances.size(),
		hipMemcpyHostToDevice));

	return true;
}

__global__ void copyPosRelease(int N, Sample* samples, float2* positions)
{
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) 
	{
		positions[i] = samples[i].pos;
		free(samples[i].components);
	}
}

void CasterCuda::loadDistances(ivhd::IGraph& graph)
{
	for (auto i = 0; i < graph.size(); i++)
	{
		auto neighbors = graph.getNeighbors(i);
		for (const auto neighbor : neighbors)
		{
			if (neighbor.type == ivhd::NeighborsType::Near)
			{ 
				distances.push_back(DistElem(neighbor.i, neighbor.j)); 
			}
			else if (neighbor.type == ivhd::NeighborsType::Random)
			{ 
				distances.push_back(DistElem(neighbor.i, neighbor.j, DistElemType::etNear, neighbor.r)); 
			}
		}
	}
}

void CasterCuda::loadPositions(ivhd::IParticleSystem& ps)
{
	auto internalPositions = ps.positions();
	for (auto i = 0; i < ps.countParticles(); i++)
	{
		positions[i].x = internalPositions[i].x;
		positions[i].y = internalPositions[i].y;
	}
}

void CasterCuda::prepare(std::vector<int>& labels)
{
	allocateInitializeDeviceMemory();
}

void CasterCuda::finish()
{
	copyResultsToHost();
}

__global__ void copyDevicePos(int N, Sample* samples, float2* positions)
{
	unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < N) 
	{
		positions[i] = samples[i].pos;
	}
}

void CasterCuda::copyPositions()
{
	copyDevicePos <<<positions.size() / 256 + 1, 256>>> (positions.size(),
		d_samples, d_positions);
	cuCall(hipMemcpy(&positions[0], d_positions,
		sizeof(float2) * positions.size(), hipMemcpyDeviceToHost));
}

bool CasterCuda::copyResultsToHost()
{
	copyPosRelease <<<positions.size() / 256 + 1, 256>>> (positions.size(),
		d_samples, d_positions);
	cuCall(hipMemcpy(&positions[0], d_positions,
		sizeof(float2) * positions.size(), hipMemcpyDeviceToHost));
	cuCall(hipFree(d_positions));
	cuCall(hipFree(d_distances));
	cuCall(hipFree(d_samples));

	return true;
}

__global__ void calculateErrors(int dstNum, DistElem* distances, Sample* samples, float* errors)
{
	for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; i < dstNum;
		i += blockDim.x * gridDim.x) 
	{
		DistElem dist = distances[i];
		float d = dist.r;
		float2 iPos = samples[dist.i].pos;
		float2 jPos = samples[dist.j].pos;
		float2 ij = { iPos.x - jPos.x, jPos.y - jPos.y };
		errors[i] = fabs(d - sqrtf(ij.x * ij.x + ij.y * ij.y));
	}
}

float CasterCuda::getError()
{
	calculateErrors <<<256, 256>>> (distances.size(), d_distances,
		d_samples, d_errors);

	thrust::device_ptr<float> err_ptr = thrust::device_pointer_cast(d_errors);
	return thrust::reduce(err_ptr, err_ptr + distances.size(), 0.0, thrust::plus<float>());
}

void CasterCuda::step(ivhd::IParticleSystem& ps, ivhd::IGraph& graph)
{
	if (!it++) 
	{
		initializeHelperVectors();
	}

	simul_step_cuda(ps, graph);

	if (it % 100 == 0) 
	{
		onError(getError());
	}

	if ((itToPosReady--) == 0) 
	{
		onPositions(positions);
	}

	if (it % 2000 == 0) 
	{
		copyPositions();
		itToPosReady = 5;
		hipDeviceSynchronize();
	}
}

void CasterCuda::prepareFromIvhdResources(ivhd::IParticleSystem& ps, ivhd::IGraph& graph)
{
	loadDistances(graph);
	loadPositions(ps);
}