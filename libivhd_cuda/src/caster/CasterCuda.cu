#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <cstring>
#include <iostream>
#include <unordered_map>
#include <thrust/execution_policy.h>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <>

#include "caster/Constants.h"
#include "caster/CasterCuda.h"

#include <IParticleSystem.h>
#include <IGraph.h>

using namespace std;
using namespace ivhd::cuda;

// initialize pos in Samples
// initialize num_components
__global__ void initializeSamples(int n, Sample *samples, float2 *positions,
    const short *sampleFreq)
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) 
    {
        Sample sample{};
        sample.pos = positions[i];
        sample.v = {0, 0};
        sample.num_components = sampleFreq[i];
        // FIXME - malloc can return NULL
        sample.components =
            (float2 *)malloc(sample.num_components * sizeof(float2));
        samples[i] = sample;
    }
}

__global__ void initializeDistances(int nDst, DistElem *distances,
    short2 *dstIndexes, Sample *samples) 
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < nDst) 
    {
        DistElem dst = distances[i];
        dst.comp1 = &samples[dst.i].components[dstIndexes[i].x];
        dst.comp2 = &samples[dst.j].components[dstIndexes[i].y];
        distances[i] = dst;
    }
}

__global__ void copyDevicePos(int N, Sample *samples, float2 *positions) 
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) 
    {
        positions[i] = samples[i].pos;
    }
}

__global__ void copyPosRelease(int N, Sample *samples, float2 *positions) 
{
    unsigned i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) 
    {
        positions[i] = samples[i].pos;
        free(samples[i].components);
    }
}

namespace ivhd { namespace cuda { namespace caster {
    void CasterCuda::initializeHelperVectors() {
        /*
        * calculate number of distances for each sample and index of each distance
        * for a given sample
        */
        auto *sampleFreq = new short[positions.size()];
        for (unsigned i = 0; i < positions.size(); i++) {
            sampleFreq[i] = 0;
        }

        auto *dstIndexes = new short2[distances.size()];

        for (unsigned i = 0; i < distances.size(); i++) {
            dstIndexes[i] = {sampleFreq[distances[i].i]++,
                             sampleFreq[distances[i].j]++};
        }

        // initialize samples
        short *d_sample_freq;
        cuCall(hipMalloc(&d_sample_freq, positions.size() * sizeof(short)));
        cuCall(hipMemcpy(d_sample_freq, sampleFreq, sizeof(short) * positions.size(),
                          hipMemcpyHostToDevice));

        delete[] sampleFreq;

        initializeSamples <<< positions.size() / 256 + 1, 256 >>> (
                positions.size(), d_samples, d_positions, d_sample_freq);
        cuCall(hipFree(d_sample_freq));

        // initialize comps in Distances in device memory
        short2 *d_dst_indexes;
        cuCall(hipMalloc(&d_dst_indexes, distances.size() * sizeof(short2)));
        cuCall(hipMemcpy(d_dst_indexes, dstIndexes,
                          sizeof(short2) * distances.size(), hipMemcpyHostToDevice));

        initializeDistances <<< distances.size() / 256 + 1, 256 >>> (
                distances.size(), d_distances, d_dst_indexes, d_samples);
        cuCall(hipFree(d_dst_indexes));
        delete[] dstIndexes;
    }

    bool CasterCuda::allocateInitializeDeviceMemory() {
        cuCall(hipMalloc(&d_positions, positions.size() * sizeof(float2)));
        cuCall(hipMalloc(&d_samples, positions.size() * sizeof(Sample)));
        cuCall(hipMalloc(&d_distances, distances.size() * sizeof(DistElem)));
        cuCall(hipMalloc(&d_errors, distances.size() * sizeof(float)));

        cuCall(hipMemcpy(d_positions, &positions[0],
                          sizeof(float2) * positions.size(), hipMemcpyHostToDevice));
        cuCall(hipMemset(d_samples, 0, positions.size() * sizeof(Sample)));
        cuCall(hipMemset(d_errors, 0, distances.size() * sizeof(float)));
        cuCall(hipMemcpy(d_distances, &distances[0],
                          sizeof(DistElem) * distances.size(),
                          hipMemcpyHostToDevice));

        return true;
    }

    void CasterCuda::initialize(ivhd::IParticleSystem &ps, ivhd::IGraph &graph) {
        auto internalPositions = ps.positions();
        auto n = ps.countParticles();

        positions.resize(n);
        internalPositions.resize(n);

        onError = [&](float err) -> void {
        };

        onPositions = [&](vector<glm::vec2> &pos) -> void {
            for (unsigned i = 0; i < pos.size(); i++) {
                internalPositions[i].x = pos[i].x;
                internalPositions[i].y = pos[i].y;
            }
        };

        // initialize positions
        for (auto i = 0; i < internalPositions.size(); i++) {
            positions[i].x = internalPositions[i].x;
            positions[i].y = internalPositions[i].y;
        }

        // initialize distances
        for (auto index = 0; index < graph.size(); index++) {
            auto neighbors = graph.getNeighbors(index);

            for (auto neighbor : neighbors) {
                distances.emplace_back(neighbor.i, neighbor.j, neighbor.type, neighbor.r);
            }
        }

        allocateInitializeDeviceMemory();
    }

    void CasterCuda::finalize() {
        copyResultsToHost();
    }

    void CasterCuda::copyPositions() {
        copyDevicePos <<< positions.size() / 256 + 1, 256 >>> (positions.size(),
                d_samples, d_positions);
        cuCall(hipMemcpy(&positions[0], d_positions,
                          sizeof(float2) * positions.size(), hipMemcpyDeviceToHost));

        for (auto i = 0; i < positions.size(); i++) {
            ivhdPositions[i].x = positions[i].x;
            ivhdPositions[i].y = positions[i].y;
        }
    }

    bool CasterCuda::copyResultsToHost() {
        copyPosRelease <<< positions.size() / 256 + 1, 256 >>> (positions.size(),
                d_samples, d_positions);
        cuCall(hipMemcpy(&positions[0], d_positions,
                          sizeof(float2) * positions.size(), hipMemcpyDeviceToHost));
        cuCall(hipFree(d_positions));
        cuCall(hipFree(d_distances));
        cuCall(hipFree(d_samples));

        return true;
    }

    __global__ void calculateErrors(int dstNum, DistElem *distances, Sample *samples, float *errors) {
        for (unsigned i = blockIdx.x * blockDim.x + threadIdx.x; i < dstNum;
             i += blockDim.x * gridDim.x) {
            DistElem dist = distances[i];
            float d = dist.r;
            float2 iPos = samples[dist.i].pos;
            float2 jPos = samples[dist.j].pos;
            float2 ij = {iPos.x - jPos.x, jPos.y - jPos.y};
            errors[i] = fabs(d - sqrtf(ij.x * ij.x + ij.y * ij.y));
        }
    }

    float CasterCuda::getError() {
        calculateErrors <<< 256, 256 >>> (distances.size(), d_distances,
                d_samples, d_errors);

        thrust::device_ptr<float> err_ptr = thrust::device_pointer_cast(d_errors);
        return thrust::reduce(err_ptr, err_ptr + distances.size(), 0.0, thrust::plus<float>());
    }

    void CasterCuda::step(ivhd::IParticleSystem &ps, ivhd::IGraph &graph) {
        if (!it++) {
            initializeHelperVectors();
        }

        simul_step_cuda();

        if (it % 100 == 0) {
            onError(getError());
        }

        if ((itToPosReady--) == 0) {

            onPositions(ivhdPositions);
        }

        if (it % 2000 == 0) {
            copyPositions();
            itToPosReady = 5;
            hipDeviceSynchronize();
        }
    }
} } }